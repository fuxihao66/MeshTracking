#include "hip/hip_runtime.h"
#define TINYOBJLOADER_IMPLEMENTATION
#include "tiny_obj_loader.h"


#include "GpuDataStructure.cuh"


#include <stdio.h>


//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

struct Vertex {
    float3 Position;
    float3 Velocity;
};


__global__ void AdvectMeshKernel(float* meshVertices, const float timeStep, const int vertexNumber) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //c[i] = a[i] + b[i];
    if (i < vertexNumber) {
        meshVertices[6 * i + 0] += timeStep * meshVertices[6 * i + 3];
        meshVertices[6 * i + 1] += timeStep * meshVertices[6 * i + 4];
        meshVertices[6 * i + 2] += timeStep * meshVertices[6 * i + 5];
    }
}

__global__ void Build3DHashTable(float* meshVertices, unsigned int* meshIndices, unsigned int triangleNum, TriangleSpatialHashMap* hashMap){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < triangleNum) {
        unsigned int vertexIndex0 = meshIndices[3 * i + 0];
        unsigned int vertexIndex1 = meshIndices[3 * i + 1];
        unsigned int vertexIndex2 = meshIndices[3 * i + 2];

        // 第0个顶点的xyz
        meshVertices[6 * vertexIndex0 + 0];
        meshVertices[6 * vertexIndex0 + 1];
        meshVertices[6 * vertexIndex0 + 2];

        // 第1个顶点的xyz
        meshVertices[6 * vertexIndex1 + 0];
        meshVertices[6 * vertexIndex1 + 1];
        meshVertices[6 * vertexIndex1 + 2];

        // 第2个顶点的xyz
        meshVertices[6 * vertexIndex2 + 0];
        meshVertices[6 * vertexIndex2 + 1];
        meshVertices[6 * vertexIndex2 + 2];

        // 计算出包含这个三角形的所有cell
        // 每个cell的entry里面加入这个三角形
        /*for each cell{
            hashMap
        }*/

        hashMap->InsertTriangle();
    }
}


__global__ void UpdateDeletionList(float* meshVertices, unsigned int* meshIndices, bool* deleteBuffer, unsigned int triangleNum, TriangleSpatialHashMap* hashMap){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < triangleNum) {
        /*unsigned int vertexIndex0 = meshIndices[3 * i + 0];
        unsigned int vertexIndex1 = meshIndices[3 * i + 1];
        unsigned int vertexIndex2 = meshIndices[3 * i + 2];*/

        //// 第0个顶点的xyz
        //meshVertices[6 * vertexIndex0 + 0];
        //meshVertices[6 * vertexIndex0 + 1];
        //meshVertices[6 * vertexIndex0 + 2];

        //// 第1个顶点的xyz
        //meshVertices[6 * vertexIndex1 + 0];
        //meshVertices[6 * vertexIndex1 + 1];
        //meshVertices[6 * vertexIndex1 + 2];

        //// 第2个顶点的xyz
        //meshVertices[6 * vertexIndex2 + 0];
        //meshVertices[6 * vertexIndex2 + 1];
        //meshVertices[6 * vertexIndex2 + 2];

        // 计算出包含这个三角形的所有cell
        // 每个cell的entry里面的所有三角形与这个三角形进行相交测试
        

        hashMap->UpdateIntersectedTriangles(meshVertices, i, deleteBuffer)

        //for (possible grid) {
        //    if grid intersect{
        //        for (triIndex in grid) {
        //            unsigned int triVertexIndex0 = meshIndices[3 * triIndex];
        //            unsigned int triVertexIndex1 = meshIndices[3 * triIndex + 1];
        //            unsigned int triVertexIndex2 = meshIndices[3 * triIndex + 2];

        //            // 根据顶点判断


        //            // 根据相交判断
        //            bool hasIntersection = TriangleIntersection(vertexIndex0, vertexIndex1, vertexIndex2, triVertexIndex0, triVertexIndex1, triVertexIndex2, meshVertices);
        //            if (hasIntersection) {
        //                atomicExch(deleteBuffer[triIndex], 1);
        //                atomicExch(deleteBuffer[i], 1);
        //            }
        //        }
        //    }
        //}
    }
}
__global__ void RemoveTopologicalNoise(float* triangleVertices, uint32_t* meshIndices, uint32_t triangleNum, bool* deletionList) {
    int triangleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (triangleIndex < triangleNum) {
        if (deletionList[triangleIndex]) {
            unsigned int vertexIndex0 = meshIndices[3 * triangleIndex + 0];
            unsigned int vertexIndex1 = meshIndices[3 * triangleIndex + 1];
            unsigned int vertexIndex2 = meshIndices[3 * triangleIndex + 2];
            // 第0个顶点的xyz
            float v0x = triangleVertices[6 * vertexIndex0 + 0];
            float v0y = triangleVertices[6 * vertexIndex0 + 1];
            float v0z = triangleVertices[6 * vertexIndex0 + 2];

            // 第1个顶点的xyz
            float v1x = triangleVertices[6 * vertexIndex1 + 0];
            float v1y = triangleVertices[6 * vertexIndex1 + 1];
            float v1z = triangleVertices[6 * vertexIndex1 + 2];

            // 第2个顶点的xyz
            float v2x = triangleVertices[6 * vertexIndex2 + 0];
            float v2y = triangleVertices[6 * vertexIndex2 + 1];
            float v2z = triangleVertices[6 * vertexIndex2 + 2];
        }
    }
}

__global__ void InsideVolumeTest() {

}
/*
__global__ void InsideVolumeKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void DeleteTriangleAndGenBoundaryKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__device__ void ManifoldEnforceThreadFunction() {
}
__global__ void ManifoldEnforcementKernel(int* c, const int* a, const int* b){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    ManifoldEnforceThreadFunction << < >> > ();   // kernel中调用kernel
}
__global__ void IdentifyHoleKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void PairHoleKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void FillHoleKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void DeleteTriangleAndUpdateBoundaryKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void ImproveMeshKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void RemoveTrianglesKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ void RemoveVerticesKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
*/


#define num_relax_iterations 5

hipError_t MeshTrack(float* cudaVertexBuffer, unsigned int* cudaIndexBuffer, unsigned int* deleteBuffer, unsigned int vertexNum, unsigned int triangleNum, TriangleSpatialHashMap* hashMap, ) {
    // 1. detect intersection
    {
        dim3 ThreadPerBlock(1024, 1, 1);
        dim3 NumBlock(ceil(triangleNum/float(ThreadPerBlock.x)), 1, 1);
        Build3DHashTable<<<NumBlock, ThreadPerBlock>>>(cudaVertexBuffer, cudaIndexBuffer, deleteBuffer, triangleNum, hashMap);
        UpdateDeletionList<<<NumBlock, ThreadPerBlock>>>();
    }

    // relaxation
    for (int i = 0; i < num_relax_iterations; i++) {
        dim3 ThreadPerBlock(1024, 1, 1);
        dim3 NumBlock(ceil(triangleNum / float(ThreadPerBlock.x)), 1, 1);
        RemoveTopologicalNoise<<<NumBlock, ThreadPerBlock>>>();
    }

    
    // 2. detect inside volume
    InsideVolumeTest<< < >> > ();
    // 3. delete and generate boundary list
    DeleteTriangleAndGenBoundaryKernel << < >> > ();
    // 4. loop
    while (1) {
        ManifoldEnforcementKernel << < >> > ();
        if () {
            break;
        }
    }
    ImproveMeshKernel << < >> > ();
    RemoveTrianglesKernel << < >> > ();
    RemoveVerticesKernel << < >> > ();
    
}


void BuildGeometry(std::vector<Vertex>& vertices, std::vector<unsigned int>& indices,
        unsigned int& indexNum, unsigned int& vertexNum){
    tinyobj::attrib_t attrib;
    std::vector<tinyobj::shape_t> shapes;
    std::vector<tinyobj::material_t> materials;
    std::string warn;
    std::string err;
    std::string filePath = "data/wooden_sphere.obj";
    bool ret = tinyobj::LoadObj(&attrib, &shapes, &materials, &warn, &err, filePath.c_str(),
        NULL, true);

    vertexNum = attrib.vertices.size() / 3 * 2;
    vertices.resize(vertexNum);
    for (size_t v = 0; v < attrib.vertices.size() / 3; v++) {
        vertices[v].Position.x = attrib.vertices[3 * v + 0];
        vertices[v].Position.y = attrib.vertices[3 * v + 1] - 0.52f;
        vertices[v].Position.z = attrib.vertices[3 * v + 2];
        vertices[v].Velocity.x = 0.0f;
        vertices[v].Velocity.y = 0.5f;
        vertices[v].Velocity.z = 0.0f;
    }

    for (size_t v = attrib.vertices.size() / 3; v < vertexNum; v++) {
        vertices[v].Position.x = attrib.vertices[3 * (v - attrib.vertices.size() / 3) + 0];
        vertices[v].Position.y = attrib.vertices[3 * (v - attrib.vertices.size() / 3) + 1] + 0.52f;
        vertices[v].Position.z = attrib.vertices[3 * (v - attrib.vertices.size() / 3) + 2];
        vertices[v].Velocity.x = 0.0f;
        vertices[v].Velocity.y = -0.5f;
        vertices[v].Velocity.z = 0.0f;
    }


    indices.resize(shapes[0].mesh.num_face_vertices.size() * 3 * 3);
    for (size_t i = 0; i < shapes.size(); i++) {

        size_t index_offset = 0;

        // For each face
        for (size_t f = 0; f < shapes[i].mesh.num_face_vertices.size(); f++) {
            size_t fnum = shapes[i].mesh.num_face_vertices[f];

            // For each vertex in the face
            for (size_t v = 0; v < fnum; v++) {
                tinyobj::index_t idx = shapes[i].mesh.indices[index_offset + v];
                indices[index_offset + v] = idx.vertex_index;
                indexNum += 1;
            }

            index_offset += fnum;
        }
    }

    for (size_t i = 0; i < indexNum; i++) {
        indices[indexNum + i] = indices[i] + attrib.vertices.size() / 3;
    }
    indexNum *= 2;
}

void 

int main()
{

    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/
    const unsigned int MaxFrameCount = 10;
    const float timeStep = 0.01f;

    const float leftBound = -1.5f;  // y
    const float rightBound = 1.5f;
    const float bottomBound = -1.5f; //z
    const float upBound = 1.5f;
    const float farBound = -1.5f; //x
    const float nearBound = 1.5f;

    const float gridSize = 0.01f;   // 需要是最长边的3倍
    const unsigned int entrySize = 100; // 每个cell最多存多少个三角形




    // 生成水滴 
    // TODO: 把cpu和gpu端资源全部封装起来
    std::vector<Vertex> vertices;
    std::vector<unsigned int> indices;
    unsigned int vertexNum = 0;
    unsigned int indexNum = 0;

    BuildGeometry(vertices, indices, vertexNum, indexNum);

    hipError_t cudaStatus = hipSetDevice(0);
    float* cudaVertexBuffer;
    unsigned int* cudaIndexBuffer;

    {//分配mesh资源    
    // TODO: 分配多两倍的空间  用于增长
        cudaStatus = hipMalloc((void**)&cudaVertexBuffer, 3 * vertexNum * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }
        cudaStatus = hipMalloc((void**)&cudaIndexBuffer, 3 * indexNum * sizeof(unsigned int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }

        cudaStatus = hipMemcpy(cudaVertexBuffer, (void*)vertices.data(), vertexNum * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
        }

        cudaStatus = hipMemcpy(cudaIndexBuffer, (void*)indices.data(), indexNum * sizeof(unsigned int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
        }
    }
    
    unsigned int* deleteBuffer;
    
    {// 分配中间变量空间
        cudaStatus = hipMalloc((void**)&deleteBuffer, indexNum * sizeof(unsigned int));  // 这里的空间是三倍三角形的数量，以防后续三角形增加
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }
    }
    IntersectionHashMap* spatialMap = new IntersectionHashMap(farBound, nearBound, leftBound, rightBound, bottomBound, upBound, gridSize, entrySize);


    // 开始处理
    for (int i = 0; i < MaxFrameCount; i++) {
        // advect mesh 
        dim3 NumBlock;
        dim3 ThreadPerBlock;
        ThreadPerBlock.x = 1024;
        ThreadPerBlock.y = 1;
        ThreadPerBlock.z = 1;
        NumBlock.x = ceil(vertexNum / float(ThreadPerBlock.x));
        NumBlock.y = 1;
        NumBlock.z = 1;
        AdvectMeshKernel<<<NumBlock, ThreadPerBlock >>>(cudaVertexBuffer, timeStep, vertexNum);
        
        // begin mesh operate
        cudaStatus = MeshTrack(cudaVertexBuffer, cudaIndexBuffer, deleteBuffer, vertexNum, indexNum / 3, spatialMap);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d\n", cudaStatus);
            return 1;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return 1;
        }
    }


   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}